#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
using namespace std;

int main(void)
{
    int device;
    int device_count;
    int numSMs;
    hipGetDeviceCount(&device_count);
    hipGetDevice(&device);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device);
    cout << "device count " << device_count << endl;
    cout << "device " << device << endl;
    cout << "numSMs " << numSMs << endl;

}
