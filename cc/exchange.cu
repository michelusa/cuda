#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <iostream>
using namespace std;


__global__ void GPU_increment_number(int* buffer, int initial)
{
    buffer[0] = 1 + initial;
}

void simple_exchange()
{
    int* buffer;
    const auto BUF_SIZE{ 1 * sizeof(int) };

    int* host_buffer = (int*)malloc(BUF_SIZE);
    *host_buffer = 99;

    hipMalloc((void**)&buffer, BUF_SIZE);
    hipMemcpy(buffer, host_buffer, BUF_SIZE, hipMemcpyHostToDevice);

    GPU_increment_number << < 1, 1 >> > (buffer, 5);

    hipDeviceSynchronize();

    hipMemcpy(host_buffer, buffer, BUF_SIZE, hipMemcpyDeviceToHost);
    printf("Incrementing result is %d \n", host_buffer[0]);

    hipFree(buffer);
    free(host_buffer);
}


