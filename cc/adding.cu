#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <iostream>
using namespace std;


__global__ void GPU_vector_add(int* left, int* right, int* result)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = left[idx] + right[idx];
}

void simple_add()
{
    const int DIM{ 1000 };
    int* left;  hipMalloc(&left, DIM * sizeof(int));
    int* right;  hipMalloc(&right, DIM * sizeof(int));
    int* result;  hipMalloc(&result, DIM * sizeof(int));

    int* main_result = new int[DIM];
    vector<int> mainv_left;  for (int i = 0; i < DIM; ++i) mainv_left.push_back(i);;
    vector<int> mainv_right;  for (int i = 0; i < DIM; ++i) mainv_right.push_back(i * 2);;

    hipMemcpy(left, mainv_left.data(), DIM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(right, mainv_right.data(), DIM * sizeof(int), hipMemcpyHostToDevice);

    hipFree(left);     hipFree(right);
    mainv_left.clear(); mainv_right.clear();

    const int thrds = 2;
    GPU_vector_add << <DIM / thrds, thrds >> > (left, right, result);


    hipMemcpy(main_result, result, DIM * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout << "simple add result:\n\t";
    for (int i = 0; i < DIM; ++i) cout << *(main_result + i) << " ";
    hipFree(result);
    delete[] main_result;
}

