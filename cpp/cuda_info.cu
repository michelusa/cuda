#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;

int main(void)
{

 '
    int device_count;
    
    hipGetDeviceCount(&device_count);
    cout << "CUDA device count " << device_count << endl;
    
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    cout << "CUDA first device name: " << device_props.name << endl;

    int driver_version;
    hipDriverGetVersion(&driver_version);
    cout << "CUDA driver version: " << driver_version << endl;

    int runtime_version;
    hipRuntimeGetVersion(&runtime_version);
    cout << "CUDA runtime version: " << runtime_version << endl;

    cout << "maxThreadsPerBlock: " << device_props.maxThreadsPerBlock << endl;    int device;

}
