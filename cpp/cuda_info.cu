#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;
__global__ void GPU_mt_info()
{
	printf("Block idx: %d | thread idx: %d\n", blockIdx.x, threadIdx.x);
}

int main()
{

	int device_count;

	hipGetDeviceCount(&device_count);
	cout << "CUDA device count " << device_count << endl;

	hipDeviceProp_t device_props;
	hipGetDeviceProperties(&device_props, 0);
	cout << "CUDA first device name: " << device_props.name << endl;

	int driver_version;
	hipDriverGetVersion(&driver_version);
	cout << "CUDA driver version: " << driver_version << endl;

	int runtime_version;
	hipRuntimeGetVersion(&runtime_version);
	cout << "CUDA runtime version: " << runtime_version << endl;

	cout << "maxThreadsPerBlock: " << device_props.maxThreadsPerBlock << endl;

	cout << "CUDA Total global mem: " << device_props.totalGlobalMem / (1048576.0) << " MB" << endl;


	hipEvent_t start;
	hipEvent_t end;
	float duration;

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	GPU_mt_info << <5, 10 >> > ();
	hipDeviceSynchronize();
	hipEventRecord(end);
	hipEventSynchronize(end);


	//https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EVENT.html#group__CUDART__EVENT_1g40159125411db92c835edb46a0989cd6
	hipEventElapsedTime(&duration, start, end);
	printf("Duration = %f ms.\n", duration);
	hipEventDestroy(start);
	hipEventDestroy(end);


	hipDeviceSynchronize();


}
