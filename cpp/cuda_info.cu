#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;
__global__ void GPU_mt_info()
{
	printf("Block idx: %d | thread idx: %d\n", blockIdx.x, threadIdx.x);
}

int main()
{

	int device_count;

	hipGetDeviceCount(&device_count);
	cout << "CUDA device count " << device_count << endl;

	hipDeviceProp_t device_props;
	hipGetDeviceProperties(&device_props, 0);
	cout << "CUDA first device name: " << device_props.name << endl;

	int driver_version;
	hipDriverGetVersion(&driver_version);
	cout << "CUDA driver version: " << driver_version << endl;

	int runtime_version;
	hipRuntimeGetVersion(&runtime_version);
	cout << "CUDA runtime version: " << runtime_version << endl;

	cout << "maxThreadsPerBlock: " << device_props.maxThreadsPerBlock << endl;

	cout << "CUDA Total global mem: " << device_props.totalGlobalMem / (1048576.0) << " MB" << endl;

	GPU_mt_info << <2, 10 >> > ();
	hipDeviceSynchronize();

}
