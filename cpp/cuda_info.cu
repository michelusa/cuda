#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;
__global__ void GPU_mt_info()
{
	printf("Block idx: %d | thread idx: %d\n", blockIdx.x, threadIdx.x);
}



__global__ void GPU_increment_number(int* buffer, int initial)
{
    buffer[0] = 1 + initial;
}

void simple_exchange()
{
    int* buffer;
    const auto BUF_SIZE{ 1 * sizeof(int) };

    int* host_buffer = (int*)malloc(BUF_SIZE);
    *host_buffer = 99;

    hipMalloc((void**)&buffer, BUF_SIZE);
    hipMemcpy(buffer, host_buffer, BUF_SIZE, hipMemcpyHostToDevice);

   GPU_increment_number <<< 1, 1 >>> (buffer, 5);
   
    hipDeviceSynchronize();

    hipMemcpy(host_buffer, buffer, BUF_SIZE, hipMemcpyDeviceToHost);
    printf("Incrementing result is %d \n", host_buffer[0]);

    hipFree(buffer);
    free(host_buffer);
}


int main()
{

	int device_count;

	hipGetDeviceCount(&device_count);
	cout << "CUDA device count " << device_count << endl;

	hipDeviceProp_t device_props;
	hipGetDeviceProperties(&device_props, 0);
	cout << "CUDA first device name: " << device_props.name << endl;

	int driver_version;
	hipDriverGetVersion(&driver_version);
	cout << "CUDA driver version: " << driver_version << endl;

	int runtime_version;
	hipRuntimeGetVersion(&runtime_version);
	cout << "CUDA runtime version: " << runtime_version << endl;

	cout << "maxThreadsPerBlock: " << device_props.maxThreadsPerBlock << endl;

	cout << "CUDA Total global mem: " << device_props.totalGlobalMem / (1048576.0) << " MB" << endl;


	hipEvent_t start;
	hipEvent_t end;
	float duration;

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	GPU_mt_info << <5, 10 >> > ();
	hipDeviceSynchronize();
	hipEventRecord(end);
	hipEventSynchronize(end);


	//https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EVENT.html#group__CUDART__EVENT_1g40159125411db92c835edb46a0989cd6
	hipEventElapsedTime(&duration, start, end);
	printf("Duration = %f ms.\n", duration);
	hipEventDestroy(start);
	hipEventDestroy(end);


	hipDeviceSynchronize();


}
