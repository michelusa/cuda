#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
using namespace std;
__global__ void GPU_mt_info()
{
	printf("Block idx: %d | thread idx: %d\n", blockIdx.x, threadIdx.x);
}



__global__ void GPU_increment_number(int* buffer, int initial)
{
    buffer[0] = 1 + initial;
}

void simple_exchange()
{
    int* buffer;
    const auto BUF_SIZE{ 1 * sizeof(int) };

    int* host_buffer = (int*)malloc(BUF_SIZE);
    *host_buffer = 99;

    hipMalloc((void**)&buffer, BUF_SIZE);
    hipMemcpy(buffer, host_buffer, BUF_SIZE, hipMemcpyHostToDevice);

   GPU_increment_number <<< 1, 1 >>> (buffer, 5);
   
    hipDeviceSynchronize();

    hipMemcpy(host_buffer, buffer, BUF_SIZE, hipMemcpyDeviceToHost);
    printf("Incrementing result is %d \n", host_buffer[0]);

    hipFree(buffer);
    free(host_buffer);
}

__global__ void GPU_vector_add(int* left, int* right, int* result)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = left[idx] + right[idx];
}

void simple_add()
{
    const int DIM{ 1000 };
    int* left;  hipMalloc(&left, DIM * sizeof(int));
    int* right;  hipMalloc(&right, DIM * sizeof(int));
    int* result;  hipMalloc(&result, DIM * sizeof(int));

    int* main_result = new int[DIM];
    vector<int> mainv_left;  for (int i = 0; i < DIM; ++i) mainv_left.push_back(i);;
    vector<int> mainv_right;  for (int i = 0; i < DIM; ++i) mainv_right.push_back(i*2);;

    hipMemcpy(left, mainv_left.data(), DIM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(right, mainv_right.data(), DIM * sizeof(int), hipMemcpyHostToDevice);

    hipFree(left);     hipFree(right);
    mainv_left.clear(); mainv_right.clear();

    const int thrds = 2;
    GPU_vector_add << <DIM/thrds, thrds >> > (left, right, result);
    hipMemcpy(main_result, result, DIM * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    cout << "simple add result:\n\t";
    for (int i = 0; i < DIM; ++i) cout << *(main_result + i) << " ";
    hipFree(result);
    delete[] main_result;
}


int main()
{

	int device_count;

	hipGetDeviceCount(&device_count);
	cout << "CUDA device count " << device_count << endl;

	hipDeviceProp_t device_props;
	hipGetDeviceProperties(&device_props, 0);
	cout << "CUDA first device name: " << device_props.name << endl;

	int driver_version;
	hipDriverGetVersion(&driver_version);
	cout << "CUDA driver version: " << driver_version << endl;

	int runtime_version;
	hipRuntimeGetVersion(&runtime_version);
	cout << "CUDA runtime version: " << runtime_version << endl;

	cout << "maxThreadsPerBlock: " << device_props.maxThreadsPerBlock << endl;

	cout << "CUDA Total global mem: " << device_props.totalGlobalMem / (1048576.0) << " MB" << endl;


	hipEvent_t start;
	hipEvent_t end;
	float duration;

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	GPU_mt_info << <5, 10 >> > ();
	hipDeviceSynchronize();
	hipEventRecord(end);
	hipEventSynchronize(end);


	//https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EVENT.html#group__CUDART__EVENT_1g40159125411db92c835edb46a0989cd6
	hipEventElapsedTime(&duration, start, end);
	printf("Duration = %f ms.\n", duration);
	hipEventDestroy(start);
	hipEventDestroy(end);


	hipDeviceSynchronize();


}
